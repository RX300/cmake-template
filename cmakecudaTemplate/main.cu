#include<iostream>
#include<hip/hip_runtime.h>

__global__ void helloFromGPU(void)
{
    unsigned int id=blockDim.x*blockIdx.x+threadIdx.x;
    printf("Hello World from GPU in thread %d!\n",id);
}
int main()
{
    helloFromGPU<<<10,10>>>();
    //将显示销毁和清理所有和设备有关的进程和资源，常放在代码结尾部分。
    hipDeviceReset();
    return 0;
}
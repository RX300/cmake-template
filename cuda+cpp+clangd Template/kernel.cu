﻿#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

// CUDA 错误检查宏 (推荐在实际项目中使用更完善的检查)
#define CHECK_CUDA_ERROR(call) do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA Error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
} while (0)

// CUDA 核函数：向量加法
__global__ void vector_add_kernel(const float* d_a, const float* d_b, float* d_c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        d_c[idx] = d_a[idx] + d_b[idx];
    }
}

// 主机端函数，负责内存分配、数据传输和核函数启动
extern "C" void run_vector_add(const std::vector<float>& h_a, const std::vector<float>& h_b, std::vector<float>& h_c) {
    int n = h_a.size();
    size_t bytes = n * sizeof(float);
    
    // 分配设备内存
    float *d_a = nullptr, *d_b = nullptr, *d_c = nullptr;
    CHECK_CUDA_ERROR(hipMalloc(&d_a, bytes));
    CHECK_CUDA_ERROR(hipMalloc(&d_b, bytes));
    CHECK_CUDA_ERROR(hipMalloc(&d_c, bytes));

    // 将数据从主机复制到设备
    CHECK_CUDA_ERROR(hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice));

    // 设置核函数启动参数
    int threads_per_block = 256;
    // 向上取整计算块数
    int blocks_per_grid = (n + threads_per_block - 1) / threads_per_block;

    std::cout << "Launching kernel with " << blocks_per_grid << " blocks and "
              << threads_per_block << " threads per block." << std::endl;

    // 启动核函数
    vector_add_kernel<<<blocks_per_grid, threads_per_block>>>(d_a, d_b, d_c, n);

    // 检查核函数启动错误 (异步错误需要后续同步才能捕获)
    CHECK_CUDA_ERROR(hipGetLastError());

    // 等待 GPU 完成计算
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    std::cout << "Kernel execution finished." << std::endl;

    // 将结果从设备复制回主机
    CHECK_CUDA_ERROR(hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost));

    // 打印结果
    std::cout << "Result: ";
    for (int i = 0; i < n; ++i) {
        std::cout << h_c[i] << " ";
    }
    std::cout << std::endl;

    // 释放设备内存
    CHECK_CUDA_ERROR(hipFree(d_a));
    CHECK_CUDA_ERROR(hipFree(d_b));
    CHECK_CUDA_ERROR(hipFree(d_c));

    std::cout << "Device memory freed." << std::endl;
}